#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "camera.h"
#include "cuda_helper.h"
#include "image_writer.h"
#include "ray_color.cuh"
#include "scene_manager.cuh"
#include "triangle_mesh.cuh"
#include "timer.h"
#include "obj_reader.cuh"
#include "model.cuh"

// TODO: Add triangle mesh intersection
__device__ Vec3 colorRay(const Ray& ray, const AABB* boxes, int num_objects) {
    // Check intersection with any box in the array
    for (int i = 0; i < num_objects; i++) {
        if (boxes->hitAABB(ray, i)) {
            // 
            switch (i % 3) {
                case 0:
                    return Vec3(1.0f, 0.0f, 0.0f); // red
                case 1:
                    return Vec3(0.0f, 1.0f, 0.0f); // green
                default:
                    return Vec3(1.0f, 1.0f, 1.0f); // white
            }
        }
    }
    // Background color (gradient from blue to white)
    return sky_bg(ray);
}

// Modified kernel to use an array of AABB objects
__global__ void renderBoxKernel(Vec3* image_buffer,
                                CUDACameraParams camera_params, AABB* boxes,
                                int num_objects) {
    int width = camera_params.pixel_width;
    int height = camera_params.pixel_height;
    Vec3 pixel00_loc = camera_params.pixel00_loc;
    Vec3 delta_u = camera_params.pixel_delta_u;
    Vec3 delta_v = camera_params.pixel_delta_v;
    Vec3 camera_origin = camera_params.center;

    // Calculate pixel coordinates
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        const int pixel_idx = row * width + col;

        // ray params
        const Vec3 pixel_center =
            pixel00_loc + (col * delta_u) + (row * delta_v);
        const Vec3 ray_direction = pixel_center - camera_origin;

        Ray ray(camera_origin, ray_direction);
        image_buffer[pixel_idx] = colorRay(ray, boxes, num_objects);
    }
}

__host__ SceneManager::SceneManager(Camera& camera, int num_objects)
    : camera(camera),
      width(camera.pixelWidth()),
      height(camera.pixelHeight()),
      num_objects(num_objects),
      h_image(nullptr),
      d_image(nullptr),
      h_aabb(nullptr),
      d_aabb(nullptr),
      h_mesh(nullptr),
      d_mesh(nullptr) {
    allocateResources();
    
    std::cout << "Scene dimensions" << width << "x" << height << std::endl;
}

// Destructor - automatically free resources (RAII)
__host__ SceneManager::~SceneManager() {
    std::cout << "Freeing resources..." << std::endl;
    freeResources();
}

// Add an AABB to the scene
__host__ void SceneManager::addAABB(float minx, float miny, float minz,
                                    float maxx, float maxy, float maxz, int obj_id) {
    // Add the AABB to the host array given the object ID
    h_aabb->h_minx[obj_id] = minx;
    h_aabb->h_miny[obj_id] = miny;
    h_aabb->h_minz[obj_id] = minz;
    h_aabb->h_maxx[obj_id] = maxx;
    h_aabb->h_maxy[obj_id] = maxy;
    h_aabb->h_maxz[obj_id] = maxz;
    h_aabb->num_obj = num_objects;

    std::cout << "Added AABB box: (" << minx << ", " << miny << ", " << minz
              << ") to (" << maxx << ", " << maxy << ", " << maxz << ")"
              << " :: Number of Objects: " << obj_id + 1<< std::endl;
}

// Add triangle mesh from OBJ file
__host__ void SceneManager::addTriangleMesh(const std::string& filename, int obj_id) {
    std::cout << "Loading triangle mesh from: " << filename << " for object ID: " << obj_id << std::endl;
    
    // Check if object ID is valid
    if (obj_id < 0 || obj_id >= num_objects) {
        std::cerr << "Invalid object ID: " << obj_id << std::endl;
        return;
    }
    
    // Use ObjReader directly to load the mesh
    ObjReader reader(filename);
    
    // Make sure the file is read successfully
    reader.readModel();
    
    // Get the triangles from the model
    const std::vector<Triangle3>& triangles = reader.parsedModel.modelTriangles;
    int numTriangles = triangles.size();
    
    if (numTriangles == 0) {
        std::cerr << "Error: No triangles found in OBJ file: " << filename << std::endl;
        return;
    }
    
    std::cout << "Successfully loaded " << numTriangles << " triangles from " << filename << std::endl;
    
    // Clear previous mesh data at this index
    h_mesh[obj_id].~TriangleMesh();
    
    // Initialize a new mesh at this index
    new(&h_mesh[obj_id]) TriangleMesh(numTriangles);
    
    // Copy triangle data to mesh
    for (int i = 0; i < numTriangles; i++) {
        // Copy vertices
        h_mesh[obj_id].h_v0x[i] = triangles[i].vertex0().x();
        h_mesh[obj_id].h_v0y[i] = triangles[i].vertex0().y();
        h_mesh[obj_id].h_v0z[i] = triangles[i].vertex0().z();
        h_mesh[obj_id].h_v1x[i] = triangles[i].vertex1().x();
        h_mesh[obj_id].h_v1y[i] = triangles[i].vertex1().y();
        h_mesh[obj_id].h_v1z[i] = triangles[i].vertex1().z();
        h_mesh[obj_id].h_v2x[i] = triangles[i].vertex2().x();
        h_mesh[obj_id].h_v2y[i] = triangles[i].vertex2().y();
        h_mesh[obj_id].h_v2z[i] = triangles[i].vertex2().z();
        // Copy normals
        h_mesh[obj_id].h_n0x[i] = triangles[i].normal0().x();
        h_mesh[obj_id].h_n0y[i] = triangles[i].normal0().y();
        h_mesh[obj_id].h_n0z[i] = triangles[i].normal0().z();
        h_mesh[obj_id].h_n1x[i] = triangles[i].normal1().x();
        h_mesh[obj_id].h_n1y[i] = triangles[i].normal1().y();
        h_mesh[obj_id].h_n1z[i] = triangles[i].normal1().z();
        h_mesh[obj_id].h_n2x[i] = triangles[i].normal2().x();
        h_mesh[obj_id].h_n2y[i] = triangles[i].normal2().y();
        h_mesh[obj_id].h_n2z[i] = triangles[i].normal2().z();
    }
    
    // Copy mesh data to device memory
    h_mesh[obj_id].meshMemcpyHtD();
   
    if (h_aabb != nullptr) {
        h_mesh->computeAABB(h_aabb, obj_id);
    }

    // Make sure the data is copied to the device
    copyToDevice();
    
    std::cout << "Mesh loaded with " << h_mesh[obj_id].numTriangles() 
              << " triangles and added to scene as object " << obj_id << std::endl;
}

// Allocate GPU resources
__host__ void SceneManager::allocateResources() {
    std::cout << "Allocating GPU resources" << std::endl;
    // Allocate host image buffer
    h_image = new Vec3[width * height];

    // Allocate device image buffer
    CUDA_CHECK(hipMalloc(&d_image, width * height * sizeof(Vec3)));

    // Allocate device AABB array
    if (num_objects > 0) {
        // Allocate a single AABB object that will handle multiple AABBs
        h_aabb = new AABB[1];
        
        // Initialize the AABB with the constructor that handles memory allocation
        *h_aabb = AABB(num_objects);
        h_aabb->mallocAABB();
        
        // Allocate GPU memory for AABB data
        h_aabb->cudaMallocAABB();
        
        // Allocate device memory for the AABB object
        CUDA_CHECK(hipMalloc(&d_aabb, sizeof(AABB)));
        
        // Allocate contiguous array of TriangleMesh objects (not pointers)
        h_mesh = new TriangleMesh[num_objects];
        
        // Allocate device memory for the array of mesh objects
        CUDA_CHECK(hipMalloc(&d_mesh, num_objects * sizeof(TriangleMesh)));
    }
}

__host__ void SceneManager::freeResources() {
    // Free host memory
    if (h_image) {
        delete[] h_image;
        h_image = nullptr;
    }
    
    if (h_aabb) {
        delete[] h_aabb;
        h_aabb = nullptr;
    }

    // Delete the triangle mesh array if it exists
    if (h_mesh) {
        delete[] h_mesh;
        h_mesh = nullptr;
    }

    // Free device memory
    if (d_mesh) {
        hipFree(d_mesh);
        d_mesh = nullptr;
    }

    if (d_image) {
        hipFree(d_image);
        d_image = nullptr;
    }

    if (d_aabb) {
        hipFree(d_aabb);
        d_aabb = nullptr;
    }
}

// Copy data to device
__host__ void SceneManager::copyToDevice() {
    // Copy AABB data to device
    if (num_objects > 0 && h_aabb != nullptr) {
        // Copy the AABB's array data to device
        h_aabb->AABBMemcpyHtD();

        // Copy the AABB object to device
        CUDA_CHECK(
            hipMemcpy(d_aabb, h_aabb, sizeof(AABB), hipMemcpyHostToDevice));
        
        // Copy the entire mesh array to device
        if (d_mesh != nullptr && h_mesh != nullptr) {
            CUDA_CHECK(hipMemcpy(d_mesh, h_mesh, num_objects * sizeof(TriangleMesh),
                                 hipMemcpyHostToDevice));
        }
    }
}

// Copy data from device
__host__ void SceneManager::copyFromDevice() {
    // Copy rendered image from device
    CUDA_CHECK(hipMemcpy(h_image, d_image, width * height * sizeof(Vec3),
                          hipMemcpyDeviceToHost));
}

// Save image to file
__host__ void SceneManager::saveImage(const char* filename) {
    writeToPPM(filename, h_image, width, height);
    std::cout << "Image saved to " << filename << std::endl;
}

__host__ void SceneManager::render() {
    // Get camera parameters from the stored reference
    CUDACameraParams camera_params = camera.CUDAparams();

    // Define grid and block dimensions
    dim3 block_dim(16, 16);
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x,
              (height + block_dim.y - 1) / block_dim.y);

    // Launch kernel
    Timer timer;
    timer.start("Rendering Scene");
    renderBoxKernel<<<grid_dim, block_dim>>>(d_image, camera_params, d_aabb,
                                      num_objects);
    hipDeviceSynchronize();
    timer.stop();
}