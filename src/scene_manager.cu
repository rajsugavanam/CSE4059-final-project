#include "hip/hip_runtime.h"
// #include <fstream>
#include <iostream>

#include "camera.h"
#include "cuda_helper.h"
#include "image_writer.h"
#include "model.cuh"
#include "obj_reader.cuh"
// #include "ray_color.cuh"
#include "scene_manager.cuh"
#include "timer.h"
#include "triangle_mesh.cuh"
#include "crt.cuh"

extern __constant__ Material c_materials[256];

// Add an AABB to the scene
__host__ void SceneManager::addAABB(float minx, float miny, float minz,
                                    float maxx, float maxy, float maxz,
                                    int obj_id) {
    // Add the AABB to the host array given the object ID
    h_aabb->h_minx[obj_id] = minx;
    h_aabb->h_miny[obj_id] = miny;
    h_aabb->h_minz[obj_id] = minz;
    h_aabb->h_maxx[obj_id] = maxx;
    h_aabb->h_maxy[obj_id] = maxy;
    h_aabb->h_maxz[obj_id] = maxz;
    h_aabb->num_obj = num_objects;

    std::cout << "Added AABB box: (" << minx << ", " << miny << ", " << minz
              << ") to (" << maxx << ", " << maxy << ", " << maxz << ")"
              << " :: Number of Objects: " << obj_id + 1 << std::endl;
}

// Add triangle mesh from OBJ file
__host__ void SceneManager::addTriangleMesh(const std::string& filename,
                                            int obj_id) {
    std::cout << "Loading triangle mesh from: " << filename
              << " for object ID: " << obj_id << std::endl;

    // Check if object ID is valid
    if (obj_id < 0 || obj_id >= num_objects) {
        std::cerr << "Invalid object ID: " << obj_id << std::endl;
        return;
    }

    // Use ObjReader directly to load the mesh
    ObjReader reader(filename);

    // Make sure the file is read successfully
    reader.readModel();

    // Get the triangles from the model
    const std::vector<Triangle3>& triangles = reader.parsedModel.modelTriangles;
    h_num_triangles[obj_id] = triangles.size();

    if (h_num_triangles[obj_id] == 0) {
        std::cerr << "Error: No triangles found in OBJ file: " << filename
                  << std::endl;
        return;
    }

    std::cout << "Successfully loaded " << h_num_triangles[obj_id] << " triangles from "
              << filename << std::endl;

    // Clear previous mesh data at this index 
    h_mesh[obj_id].~TriangleMesh();

    // Initialize a new mesh at this index
    new (&h_mesh[obj_id]) TriangleMesh(h_num_triangles[obj_id]);

    // Copy triangle data to mes
    for (int i = 0; i < h_num_triangles[obj_id]; i++) {
        // Copy vertices
        h_mesh[obj_id].h_v0x[i] = triangles[i].vertex0().x();
        h_mesh[obj_id].h_v0y[i] = triangles[i].vertex0().y();
        h_mesh[obj_id].h_v0z[i] = triangles[i].vertex0().z();
        h_mesh[obj_id].h_v1x[i] = triangles[i].vertex1().x();
        h_mesh[obj_id].h_v1y[i] = triangles[i].vertex1().y();
        h_mesh[obj_id].h_v1z[i] = triangles[i].vertex1().z();
        h_mesh[obj_id].h_v2x[i] = triangles[i].vertex2().x();
        h_mesh[obj_id].h_v2y[i] = triangles[i].vertex2().y();
        h_mesh[obj_id].h_v2z[i] = triangles[i].vertex2().z();
        // Copy normals
        h_mesh[obj_id].h_n0x[i] = triangles[i].normal0().x();
        h_mesh[obj_id].h_n0y[i] = triangles[i].normal0().y();
        h_mesh[obj_id].h_n0z[i] = triangles[i].normal0().z();
        h_mesh[obj_id].h_n1x[i] = triangles[i].normal1().x();
        h_mesh[obj_id].h_n1y[i] = triangles[i].normal1().y();
        h_mesh[obj_id].h_n1z[i] = triangles[i].normal1().z();
        h_mesh[obj_id].h_n2x[i] = triangles[i].normal2().x();
        h_mesh[obj_id].h_n2y[i] = triangles[i].normal2().y();
        h_mesh[obj_id].h_n2z[i] = triangles[i].normal2().z();
    }

    // Copy mesh data to device memory
    h_mesh[obj_id].meshMemcpyHtD();

    if (h_aabb != nullptr) {
        h_mesh->computeAABB(h_aabb, obj_id);
    }

    // Make sure the data is copied to the device
    copyToDevice();

    std::cout << "Mesh loaded with " << h_mesh[obj_id].numTriangles()
              << " triangles and added to scene as object " << obj_id
              << std::endl;
}

__host__ void SceneManager::addTriangleMeshColor(const std::string& filename, float3 albedo,
                                            int obj_id) {
    addTriangleMesh(filename, obj_id);
    materials[obj_id].albedo = albedo;
}

// Allocate GPU resources
__host__ void SceneManager::allocateResources() {
    std::cout << "Allocating GPU resources" << std::endl;
    // Allocate host image buffer
    h_image = new Vec3[width * height];

    // Allocate device image buffer
    CUDA_CHECK(hipMalloc(&d_image, width * height * sizeof(Vec3)));

    // Allocate device AABB array
    if (num_objects > 0) {
        // Allocate a single AABB object that will handle multiple AABBs
        h_aabb = new AABB[1];

        // Initialize the AABB with the constructor that handles memory
        // allocation
        *h_aabb = AABB(num_objects);
        h_aabb->mallocAABB();

        // Allocate GPU memory for AABB data
        h_aabb->cudaMallocAABB();

        // Allocate device memory for the AABB object
        CUDA_CHECK(hipMalloc(&d_aabb, sizeof(AABB)));

        // Allocate contiguous array of TriangleMesh objects (not pointers)
        h_mesh = new TriangleMesh[num_objects];

        // Allocate device memory for the array of mesh objects
        CUDA_CHECK(hipMalloc(&d_mesh, num_objects * sizeof(TriangleMesh)));

        // Allocate memory for the number of triangles in each mesh
        h_num_triangles = new int[num_objects];
        CUDA_CHECK(hipMalloc(&d_num_triangles, num_objects * sizeof(int)));
    }
}

__host__ void SceneManager::freeResources() {
    // Free host memory
    if (h_image) {
        delete[] h_image;
        h_image = nullptr;
    }

    if (h_aabb) {
        delete[] h_aabb;
        h_aabb = nullptr;
    }

    // Delete the triangle mesh array if it exists
    if (h_mesh) {
        delete[] h_mesh;
        h_mesh = nullptr;
    }

    // Delete the number of triangles array if it exists
    if (h_num_triangles) {
        delete[] h_num_triangles;
        h_num_triangles = nullptr;
    }

    // Free device memory
    if (d_mesh) {
        hipFree(d_mesh);
        d_mesh = nullptr;
    }

    if (d_image) {
        hipFree(d_image);
        d_image = nullptr;
    }

    if (d_aabb) {
        hipFree(d_aabb);
        d_aabb = nullptr;
    }

    if (d_num_triangles) {
        hipFree(d_num_triangles);
        d_num_triangles = nullptr;
    }
}

// Copy data to device
__host__ void SceneManager::copyToDevice() {
    // Copy AABB data to device
    if (num_objects > 0 && h_aabb != nullptr) {
        // Copy the AABB's array data to device
        h_aabb->AABBMemcpyHtD();

        // Copy the AABB object to device
        CUDA_CHECK(
            hipMemcpy(d_aabb, h_aabb, sizeof(AABB), hipMemcpyHostToDevice));

        // Copy the entire mesh array to device
        if (d_mesh != nullptr && h_mesh != nullptr) {
            CUDA_CHECK(hipMemcpy(d_mesh, h_mesh,
                                  num_objects * sizeof(TriangleMesh),
                                  hipMemcpyHostToDevice));
        }

        // Copy the number of triangles to device
        if (d_num_triangles != nullptr && h_num_triangles != nullptr) {
            CUDA_CHECK(hipMemcpy(d_num_triangles, h_num_triangles,
                                  num_objects * sizeof(int),
                                  hipMemcpyHostToDevice));
        }
    }
}

// Copy data from device
__host__ void SceneManager::copyFromDevice() {
    // Copy rendered image from device
    CUDA_CHECK(hipMemcpy(h_image, d_image, width * height * sizeof(Vec3),
                          hipMemcpyDeviceToHost));
}

// Save image to file
__host__ void SceneManager::saveImage(const char* filename) {
    writeToPPM(filename, h_image, width, height);
    std::cout << "Image saved to " << filename << std::endl;
}

__host__ void SceneManager::renderBox() {
    // Get camera parameters from the stored reference
    CUDACameraParams camera_params = camera.CUDAparams();

    // Define grid and block dimensions
    dim3 block_dim(16, 16);
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x,
                  (height + block_dim.y - 1) / block_dim.y);

    // Launch kernel
    Timer timer;
    timer.start("Rendering Scene");
    renderBoxKernel<<<grid_dim, block_dim>>>(d_image, camera_params, d_aabb,
                                             num_objects);
    hipDeviceSynchronize();
    timer.stop();
}

__host__ void SceneManager::renderMesh() {
    // Get camera parameters from the stored reference
    CUDACameraParams camera_params = camera.CUDAparams();

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_materials), materials,
        sizeof(Material) * num_objects));

    // Define grid and block dimensions
    dim3 block_dim(32, 4); // Adjusted for better warp scheduling / occupancy
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x,
                  (height + block_dim.y - 1) / block_dim.y);

    Timer timer;
    timer.start("Rendering Scene");
    renderMeshKernel<<<grid_dim, block_dim>>>(d_image, d_aabb, d_mesh, 
                                              num_objects, d_num_triangles, camera_params);
    hipDeviceSynchronize();
    timer.stop();
    CUDA_CHECK(hipGetLastError());
}
