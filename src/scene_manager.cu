#include "hip/hip_runtime.h"
// #include <fstream>
#include <iostream>

#include "camera.h"
#include "cuda_helper.h"
#include "image_writer.h"
#include "model.cuh"
#include "obj_reader.cuh"
#include "ray_color.cuh"
#include "scene_manager.cuh"
#include "timer.h"
#include "triangle_mesh.cuh"

// Color ray based on hit triangle
// FIRST CHECK AABB, THEN CHECK TRIANGLE
__device__ Vec3 colorRayTriangle(const Ray& ray, const AABB* boxes, const TriangleMesh* mesh,
                                  int num_objects) {
    // Check intersection with any triangle in the array
    for (int i = 0; i < num_objects; i++) {
        if (boxes->hitAABB(ray, i)) {
            // Check intersection with the triangle
            // if (mesh->hitTriangle(ray, i)) {
            //     // Return color based on triangle ID
            //     switch (i % 3) {
            //         case 0:
            //             return Vec3(1.0f, 0.0f, 0.0f);  // red
            //         case 1:
            //             return Vec3(0.0f, 1.0f, 0.0f);  // green
            //         default:
            //             return Vec3(0.0f, 0.0f, 1.0f);  // blue
            //     }
            // }
            return Vec3(1.0f, 0.0f, 0.0f);  // No hit
        } else {
            return sky_bg(ray);
        }
    }

    return Vec3(0.0f, 0.0f, 1.0f);  // No hit

}

// TODO: Add triangle mesh intersection
__device__ Vec3 colorRayBox(const Ray& ray, const AABB* boxes,
                            int num_objects) {
    // Check intersection with any box in the array
    for (int i = 0; i < num_objects; i++) {
        if (boxes->hitAABB(ray, i)) {
            // 
            switch (i % 3) {
                case 0:
                    return Vec3(1.0f, 0.0f, 0.0f);  // red
                case 1:
                    return Vec3(0.0f, 1.0f, 0.0f);  // green
                default:
                    return Vec3(0.0f, 0.0f, 1.0f);  // white
            }
        }
    }
    // Background color (gradient from blue to white)
    return sky_bg(ray);
}

// Modified kernel to use an array of AABB objects
__global__ void renderBoxKernel(Vec3* image_buffer,
                                CUDACameraParams camera_params, AABB* boxes,
                                int num_objects) {
    int width = camera_params.pixel_width;
    int height = camera_params.pixel_height;
    Vec3 pixel00_loc = camera_params.pixel00_loc;
    Vec3 delta_u = camera_params.pixel_delta_u;
    Vec3 delta_v = camera_params.pixel_delta_v;
    Vec3 camera_origin = camera_params.center;

    // Calculate pixel coordinates
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        const int pixel_idx = row * width + col;

        // ray params
        const Vec3 pixel_center =
            pixel00_loc + (col * delta_u) + (row * delta_v);
        const Vec3 ray_direction = pixel_center - camera_origin;

        Ray ray(camera_origin, ray_direction);
        image_buffer[pixel_idx] = colorRayBox(ray, boxes, num_objects);
    }
}

__global__ void renderMeshKernel(Vec3* image_buffer, AABB* boxes, TriangleMesh* meshes,
                                 const int num_objects, const int* __restrict__ num_triangles,
                                 const CUDACameraParams camera_params) {
    
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < camera_params.pixel_width && row < camera_params.pixel_height) {
        const int pixel_idx = row * camera_params.pixel_width + col;

        // ray params
        const Vec3 pixel_center = camera_params.pixel00_loc +
                                  (col * camera_params.pixel_delta_u) +
                                  (row * camera_params.pixel_delta_v);
        const Vec3 ray_direction = pixel_center - camera_params.center;

        Ray ray(camera_params.center, ray_direction);
        image_buffer[pixel_idx] = colorRayBox(ray, boxes, num_objects);
    }
}

__host__ SceneManager::SceneManager(Camera& camera, int num_objects)
    : camera(camera),
      width(camera.pixelWidth()),
      height(camera.pixelHeight()),
      num_objects(num_objects),
      h_image(nullptr),
      d_image(nullptr),
      h_aabb(nullptr),
      d_aabb(nullptr),
      h_num_triangles(nullptr),
      d_num_triangles(nullptr),
      h_mesh(nullptr),
      d_mesh(nullptr) {
    allocateResources();

    std::cout << "Scene dimensions: " << width << "x" << height << std::endl;
}

// Destructor - automatically free resources (RAII)
__host__ SceneManager::~SceneManager() {
    std::cout << "Freeing resources..." << std::endl;
    freeResources();
}

// Add an AABB to the scene
__host__ void SceneManager::addAABB(float minx, float miny, float minz,
                                    float maxx, float maxy, float maxz,
                                    int obj_id) {
    // Add the AABB to the host array given the object ID
    h_aabb->h_minx[obj_id] = minx;
    h_aabb->h_miny[obj_id] = miny;
    h_aabb->h_minz[obj_id] = minz;
    h_aabb->h_maxx[obj_id] = maxx;
    h_aabb->h_maxy[obj_id] = maxy;
    h_aabb->h_maxz[obj_id] = maxz;
    h_aabb->num_obj = num_objects;

    std::cout << "Added AABB box: (" << minx << ", " << miny << ", " << minz
              << ") to (" << maxx << ", " << maxy << ", " << maxz << ")"
              << " :: Number of Objects: " << obj_id + 1 << std::endl;
}

// Add triangle mesh from OBJ file
__host__ void SceneManager::addTriangleMesh(const std::string& filename,
                                            int obj_id) {
    std::cout << "Loading triangle mesh from: " << filename
              << " for object ID: " << obj_id << std::endl;

    // Check if object ID is valid
    if (obj_id < 0 || obj_id >= num_objects) {
        std::cerr << "Invalid object ID: " << obj_id << std::endl;
        return;
    }

    // Use ObjReader directly to load the mesh
    ObjReader reader(filename);

    // Make sure the file is read successfully
    reader.readModel();

    // Get the triangles from the model
    const std::vector<Triangle3>& triangles = reader.parsedModel.modelTriangles;
    h_num_triangles[obj_id] = triangles.size();

    if (h_num_triangles[obj_id] == 0) {
        std::cerr << "Error: No triangles found in OBJ file: " << filename
                  << std::endl;
        return;
    }

    std::cout << "Successfully loaded " << h_num_triangles[obj_id] << " triangles from "
              << filename << std::endl;

    // Clear previous mesh data at this index 
    h_mesh[obj_id].~TriangleMesh();

    // Initialize a new mesh at this index
    new (&h_mesh[obj_id]) TriangleMesh(h_num_triangles[obj_id]);

    // Copy triangle data to mes
    for (int i = 0; i < h_num_triangles[obj_id]; i++) {
        // Copy vertices
        h_mesh[obj_id].h_v0x[i] = triangles[i].vertex0().x();
        h_mesh[obj_id].h_v0y[i] = triangles[i].vertex0().y();
        h_mesh[obj_id].h_v0z[i] = triangles[i].vertex0().z();
        h_mesh[obj_id].h_v1x[i] = triangles[i].vertex1().x();
        h_mesh[obj_id].h_v1y[i] = triangles[i].vertex1().y();
        h_mesh[obj_id].h_v1z[i] = triangles[i].vertex1().z();
        h_mesh[obj_id].h_v2x[i] = triangles[i].vertex2().x();
        h_mesh[obj_id].h_v2y[i] = triangles[i].vertex2().y();
        h_mesh[obj_id].h_v2z[i] = triangles[i].vertex2().z();
        // Copy normals
        h_mesh[obj_id].h_n0x[i] = triangles[i].normal0().x();
        h_mesh[obj_id].h_n0y[i] = triangles[i].normal0().y();
        h_mesh[obj_id].h_n0z[i] = triangles[i].normal0().z();
        h_mesh[obj_id].h_n1x[i] = triangles[i].normal1().x();
        h_mesh[obj_id].h_n1y[i] = triangles[i].normal1().y();
        h_mesh[obj_id].h_n1z[i] = triangles[i].normal1().z();
        h_mesh[obj_id].h_n2x[i] = triangles[i].normal2().x();
        h_mesh[obj_id].h_n2y[i] = triangles[i].normal2().y();
        h_mesh[obj_id].h_n2z[i] = triangles[i].normal2().z();
    }

    // Copy mesh data to device memory
    h_mesh[obj_id].meshMemcpyHtD();

    if (h_aabb != nullptr) {
        h_mesh->computeAABB(h_aabb, obj_id);
    }

    // Make sure the data is copied to the device
    copyToDevice();

    std::cout << "Mesh loaded with " << h_mesh[obj_id].numTriangles()
              << " triangles and added to scene as object " << obj_id
              << std::endl;
}

// Allocate GPU resources
__host__ void SceneManager::allocateResources() {
    std::cout << "Allocating GPU resources" << std::endl;
    // Allocate host image buffer
    h_image = new Vec3[width * height];

    // Allocate device image buffer
    CUDA_CHECK(hipMalloc(&d_image, width * height * sizeof(Vec3)));

    // Allocate device AABB array
    if (num_objects > 0) {
        // Allocate a single AABB object that will handle multiple AABBs
        h_aabb = new AABB[1];

        // Initialize the AABB with the constructor that handles memory
        // allocation
        *h_aabb = AABB(num_objects);
        h_aabb->mallocAABB();

        // Allocate GPU memory for AABB data
        h_aabb->cudaMallocAABB();

        // Allocate device memory for the AABB object
        CUDA_CHECK(hipMalloc(&d_aabb, sizeof(AABB)));

        // Allocate contiguous array of TriangleMesh objects (not pointers)
        h_mesh = new TriangleMesh[num_objects];

        // Allocate device memory for the array of mesh objects
        CUDA_CHECK(hipMalloc(&d_mesh, num_objects * sizeof(TriangleMesh)));

        // Allocate memory for the number of triangles in each mesh
        h_num_triangles = new int[num_objects];
        CUDA_CHECK(hipMalloc(&d_num_triangles, num_objects * sizeof(int)));
    }
}

__host__ void SceneManager::freeResources() {
    // Free host memory
    if (h_image) {
        delete[] h_image;
        h_image = nullptr;
    }

    if (h_aabb) {
        delete[] h_aabb;
        h_aabb = nullptr;
    }

    // Delete the triangle mesh array if it exists
    if (h_mesh) {
        delete[] h_mesh;
        h_mesh = nullptr;
    }

    // Delete the number of triangles array if it exists
    if (h_num_triangles) {
        delete[] h_num_triangles;
        h_num_triangles = nullptr;
    }

    // Free device memory
    if (d_mesh) {
        hipFree(d_mesh);
        d_mesh = nullptr;
    }

    if (d_image) {
        hipFree(d_image);
        d_image = nullptr;
    }

    if (d_aabb) {
        hipFree(d_aabb);
        d_aabb = nullptr;
    }

    if (d_num_triangles) {
        hipFree(d_num_triangles);
        d_num_triangles = nullptr;
    }
}

// Copy data to device
__host__ void SceneManager::copyToDevice() {
    // Copy AABB data to device
    if (num_objects > 0 && h_aabb != nullptr) {
        // Copy the AABB's array data to device
        h_aabb->AABBMemcpyHtD();

        // Copy the AABB object to device
        CUDA_CHECK(
            hipMemcpy(d_aabb, h_aabb, sizeof(AABB), hipMemcpyHostToDevice));

        // Copy the entire mesh array to device
        if (d_mesh != nullptr && h_mesh != nullptr) {
            CUDA_CHECK(hipMemcpy(d_mesh, h_mesh,
                                  num_objects * sizeof(TriangleMesh),
                                  hipMemcpyHostToDevice));
        }

        // Copy the number of triangles to device
        if (d_num_triangles != nullptr && h_num_triangles != nullptr) {
            CUDA_CHECK(hipMemcpy(d_num_triangles, h_num_triangles,
                                  num_objects * sizeof(int),
                                  hipMemcpyHostToDevice));
        }
    }
}

// Copy data from device
__host__ void SceneManager::copyFromDevice() {
    // Copy rendered image from device
    CUDA_CHECK(hipMemcpy(h_image, d_image, width * height * sizeof(Vec3),
                          hipMemcpyDeviceToHost));
}

// Save image to file
__host__ void SceneManager::saveImage(const char* filename) {
    writeToPPM(filename, h_image, width, height);
    std::cout << "Image saved to " << filename << std::endl;
}

__host__ void SceneManager::renderBox() {
    // Get camera parameters from the stored reference
    CUDACameraParams camera_params = camera.CUDAparams();

    // Define grid and block dimensions
    dim3 block_dim(16, 16);
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x,
                  (height + block_dim.y - 1) / block_dim.y);

    // Launch kernel
    Timer timer;
    timer.start("Rendering Scene");
    renderBoxKernel<<<grid_dim, block_dim>>>(d_image, camera_params, d_aabb,
                                             num_objects);
    hipDeviceSynchronize();
    timer.stop();
}

__host__ void SceneManager::renderMesh() {
    // Get camera parameters from the stored reference
    CUDACameraParams camera_params = camera.CUDAparams();

    // Define grid and block dimensions
    dim3 block_dim(16, 16);
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x,
                  (height + block_dim.y - 1) / block_dim.y);

//     // Launch kernel
//     std::cout << "Render Parameters: "
//               << "Width: " << width << ", Height: " << height
//               << ", Num Objects: " << num_objects << std::endl;
//     std::cout << "Camera Parameters: "
//               << "Pixel Width: " << camera_params.pixel_width
//               << ", Pixel Height: " << camera_params.pixel_height
//               << ", Pixel00 Location: " << camera_params.pixel00_loc
//               << ", Delta U: " << camera_params.pixel_delta_u
//               << ", Delta V: " << camera_params.pixel_delta_v
//               << ", Camera Center: " << camera_params.center << std::endl;
              
//    for (int i = 0; i < num_objects; i++) {
//         std::cout << "AABB Object ID: " << i << std::endl;
//         std::cout << "AABB Parameters: "
//                   << "MinX: " << h_aabb->h_minx[i]
//                   << ", MinY: " << h_aabb->h_miny[i]
//                   << ", MinZ: " << h_aabb->h_minz[i]
//                   << ", MaxX: " << h_aabb->h_maxx[i]
//                   << ", MaxY: " << h_aabb->h_maxy[i]
//                   << ", MaxZ: " << h_aabb->h_maxz[i] << std::endl;
//     }
//     std::cout << "Triangle Mesh Parameters: "
//               << "Num Triangles: " << h_mesh[0].num_triangles[obj_id]() << std::endl;
//     // Print the first few triangles for debugging
//     const int num_print_tri = std::min(5, h_mesh[0].num_triangles[obj_id]());
//     std::cout << "Printing first " << num_print_tri << " triangles:" << std::endl;
//     for (int i = 0; i < num_print_tri; i++) {
//         std::cout << "Triangle " << i << ": "
//                   << "V0: (" << h_mesh[0].h_v0x[i] << ", " << h_mesh[0].h_v0y[i]
//                   << ", " << h_mesh[0].h_v0z[i] << ")"
//                   << ", V1: (" << h_mesh[0].h_v1x[i] << ", " << h_mesh[0].h_v1y[i]
//                   << ", " << h_mesh[0].h_v1z[i] << ")"
//                   << ", V2: (" << h_mesh[0].h_v2x[i] << ", " << h_mesh[0].h_v2y[i]
//                   << ", " << h_mesh[0].h_v2z[i] << ")" << std::endl;
//     }
    Timer timer;
    timer.start("Rendering Scene");
    renderMeshKernel<<<grid_dim, block_dim>>>(d_image, d_aabb, d_mesh, 
                                              num_objects, d_num_triangles, camera_params);
    hipDeviceSynchronize();
    timer.stop();
    CUDA_CHECK(hipGetLastError());
} 
