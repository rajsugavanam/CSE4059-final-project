#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <thread>

#include "camera.h"
#include "crt.cuh"
#include "cuda_helper.h"
#include "image_writer.h"
#include "model.cuh"
#include "obj_reader.cuh"
#include "scene_manager.cuh"
#include "timer.h"
#include "triangle_mesh.cuh"
#include "util/cb_light_spectrum.h"
#include "util/cb_spectrum.h"
#include "util/cie_spectrum.h"

// Define the constant memory variables here once (extern everywhere else)
__constant__ Material c_materials[256];
__constant__ float3 c_cieXYZ[301];
__constant__ float3 c_cieXYZ_to_sRGB[3];
__constant__ float c_white_reflectance[301];
__constant__ float c_green_reflectance[301];
__constant__ float c_red_reflectance[301];
__constant__ float c_light_emission[301];
__constant__ float c_light_reflectance[301];

__host__ SceneManager::SceneManager(Camera& camera, int num_objects)
    : camera(camera),
      width(camera.pixelWidth()),
      height(camera.pixelHeight()),
      num_objects(num_objects),
      h_image(nullptr),
      d_image(nullptr),
      h_aabb(nullptr),
      d_aabb(nullptr),
      h_num_triangles(nullptr),
      d_num_triangles(nullptr),
      h_mesh(nullptr),
      d_mesh(nullptr) {
    allocateResources();

    std::cout << "Scene dimensions: " << width << "x" << height << std::endl;
}

// Destructor - automatically free resources (RAII)
__host__ SceneManager::~SceneManager() {
    std::cout << "Freeing resources..." << std::endl;
    freeResources();
}

// Initialize spectral data in constant memory
__host__ void SceneManager::initializeSpectra() {
    // std::cout << "Initializing spectral data..." << std::endl;

    // Copy spectral data directly from the header file constants to GPU
    // constant memory
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(
        c_white_reflectance), WHITE_REFLECTANCE_SPECTRUM, 301 * sizeof(float)));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(
        c_green_reflectance), GREEN_REFLECTANCE_SPECTRUM, 301 * sizeof(float)));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_red_reflectance), RED_REFLECTANCE_SPECTRUM,
                                  301 * sizeof(float)));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_light_emission), LIGHT_EMISSION_SPECTRUM,
                                  301 * sizeof(float)));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(
        c_light_reflectance), LIGHT_REFLECTANCE_SPECTRUM, 301 * sizeof(float)));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_cieXYZ), CIE_COLOR_MATCHING_FUNCTIONS,
                                  301 * sizeof(float3)));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_cieXYZ_to_sRGB), CIE_XYZ_TO_SRGB,
                                  3 * sizeof(float3)));

    // std::cout << "Spectral data initialization complete." << std::endl;
}

// Add an AABB to the scene
__host__ void SceneManager::addAABB(float minx, float miny, float minz,
                                    float maxx, float maxy, float maxz,
                                    int obj_id) {
    // Add the AABB to the host array given the object ID
    h_aabb->h_minx[obj_id] = minx;
    h_aabb->h_miny[obj_id] = miny;
    h_aabb->h_minz[obj_id] = minz;
    h_aabb->h_maxx[obj_id] = maxx;
    h_aabb->h_maxy[obj_id] = maxy;
    h_aabb->h_maxz[obj_id] = maxz;
    h_aabb->num_obj = num_objects;

    // std::cout << "Added AABB box: (" << minx << ", " << miny << ", " << minz
    //           << ") to (" << maxx << ", " << maxy << ", " << maxz << ")"
    //           << " :: Number of Objects: " << obj_id + 1 << std::endl;
}

// Add triangle mesh from OBJ file
__host__ void SceneManager::addTriangleMesh(const std::string& filename,
                                            int obj_id) {
    // std::cout << "Loading triangle mesh from: " << filename
    //           << " for object ID: " << obj_id << std::endl;

    // Check if object ID is valid
    if (obj_id < 0 || obj_id >= num_objects) {
        std::cerr << "Invalid object ID: " << obj_id << std::endl;
        return;
    }

    // Use ObjReader directly to load the mesh
    ObjReader reader(filename);

    // Make sure the file is read successfully
    reader.readModel();

    // Get the triangles from the model
    const std::vector<Triangle3>& triangles = reader.parsedModel.modelTriangles;
    h_num_triangles[obj_id] = triangles.size();

    if (h_num_triangles[obj_id] == 0) {
        std::cerr << "Error: No triangles found in OBJ file: " << filename
                  << std::endl;
        return;
    }

    // std::cout << "Successfully loaded " << h_num_triangles[obj_id]
    //           << " triangles from " << filename << std::endl;

    // Clear previous mesh data at this index
    h_mesh[obj_id].~TriangleMesh();

    // Initialize a new mesh at this index
    new (&h_mesh[obj_id]) TriangleMesh(h_num_triangles[obj_id]);

    // Copy triangle data to mes
    for (int i = 0; i < h_num_triangles[obj_id]; i++) {
        // Copy vertices
        h_mesh[obj_id].h_v0x[i] = triangles[i].vertex0().x();
        h_mesh[obj_id].h_v0y[i] = triangles[i].vertex0().y();
        h_mesh[obj_id].h_v0z[i] = triangles[i].vertex0().z();
        h_mesh[obj_id].h_v1x[i] = triangles[i].vertex1().x();
        h_mesh[obj_id].h_v1y[i] = triangles[i].vertex1().y();
        h_mesh[obj_id].h_v1z[i] = triangles[i].vertex1().z();
        h_mesh[obj_id].h_v2x[i] = triangles[i].vertex2().x();
        h_mesh[obj_id].h_v2y[i] = triangles[i].vertex2().y();
        h_mesh[obj_id].h_v2z[i] = triangles[i].vertex2().z();
        // Copy normals
        h_mesh[obj_id].h_n0x[i] = triangles[i].normal0().x();
        h_mesh[obj_id].h_n0y[i] = triangles[i].normal0().y();
        h_mesh[obj_id].h_n0z[i] = triangles[i].normal0().z();
        h_mesh[obj_id].h_n1x[i] = triangles[i].normal1().x();
        h_mesh[obj_id].h_n1y[i] = triangles[i].normal1().y();
        h_mesh[obj_id].h_n1z[i] = triangles[i].normal1().z();
        h_mesh[obj_id].h_n2x[i] = triangles[i].normal2().x();
        h_mesh[obj_id].h_n2y[i] = triangles[i].normal2().y();
        h_mesh[obj_id].h_n2z[i] = triangles[i].normal2().z();
    }

    // Copy mesh data to device memory
    h_mesh[obj_id].meshMemcpyHtD();

    if (h_aabb != nullptr) {
        h_mesh->computeAABB(h_aabb, obj_id);
    }

    // Make sure the data is copied to the device
    copyToDevice();

    // std::cout << "Mesh loaded with " << h_mesh[obj_id].numTriangles()
    //           << " triangles and added to scene as object " << obj_id
    //           << std::endl;
}

// Old function to add triangle mesh with color
__host__ void SceneManager::addTriangleMeshColor(const std::string& filename,
                                                 float3 albedo, int obj_id) {
    addTriangleMesh(filename, obj_id);
    materials[obj_id].albedo = albedo;
}

// New function to add triangle mesh with spectral properties
__host__ void SceneManager::addTriangleMeshSpectrum(const std::string& filename,
                                                    int spectral_reflectance_id,
                                                    int spectral_emission_id,
                                                    bool is_emissive,
                                                    int obj_id) {
    // Add the mesh geometry first
    addTriangleMesh(filename, obj_id);

    // Then set the spectral material properties
    materials[obj_id].spectral_reflectance_id = spectral_reflectance_id;
    materials[obj_id].spectral_emission_id = spectral_emission_id;
    materials[obj_id].is_emissive = is_emissive;
    materials[obj_id].type =
        is_emissive ? MaterialType::EMISSIVE : MaterialType::DIFFUSE;
    // For emissive materials, set the albedo to be high for regular rendering
    // path
    if (is_emissive) {
        materials[obj_id].albedo = make_float3(10.0f, 10.0f, 10.0f);
    }
}

// Allocate GPU resources
__host__ void SceneManager::allocateResources() {
    // std::cout << "Allocating GPU resources" << std::endl;
    // Allocate host image buffer
    h_image = new Vec3[width * height];

    // Allocate device image buffer
    CUDA_CHECK(hipMalloc(&d_image, width * height * sizeof(Vec3)));

    // Allocate device AABB array
    if (num_objects > 0) {
        // Allocate a single AABB object that will handle multiple AABBs
        h_aabb = new AABB[1];

        // Initialize the AABB with the constructor that handles memory
        // allocation
        *h_aabb = AABB(num_objects);
        h_aabb->mallocAABB();

        // Allocate GPU memory for AABB data
        h_aabb->cudaMallocAABB();

        // Allocate device memory for the AABB object
        CUDA_CHECK(hipMalloc(&d_aabb, sizeof(AABB)));

        // Allocate contiguous array of TriangleMesh objects (not pointers)
        h_mesh = new TriangleMesh[num_objects];

        // Allocate device memory for the array of mesh objects
        CUDA_CHECK(hipMalloc(&d_mesh, num_objects * sizeof(TriangleMesh)));

        // Allocate memory for the number of triangles in each mesh
        h_num_triangles = new int[num_objects];
        CUDA_CHECK(hipMalloc(&d_num_triangles, num_objects * sizeof(int)));
    }
}

__host__ void SceneManager::freeResources() {
    // Free host memory
    if (h_image) {
        delete[] h_image;
        h_image = nullptr;
    }

    if (h_aabb) {
        delete[] h_aabb;
        h_aabb = nullptr;
    }

    // Delete the triangle mesh array if it exists
    if (h_mesh) {
        delete[] h_mesh;
        h_mesh = nullptr;
    }

    // Delete the number of triangles array if it exists
    if (h_num_triangles) {
        delete[] h_num_triangles;
        h_num_triangles = nullptr;
    }

    // Free device memory
    if (d_mesh) {
        hipFree(d_mesh);
        d_mesh = nullptr;
    }

    if (d_image) {
        hipFree(d_image);
        d_image = nullptr;
    }

    if (d_aabb) {
        hipFree(d_aabb);
        d_aabb = nullptr;
    }

    if (d_num_triangles) {
        hipFree(d_num_triangles);
        d_num_triangles = nullptr;
    }
}

// Copy data to device
__host__ void SceneManager::copyToDevice() {
    // Copy AABB data to device
    if (num_objects > 0 && h_aabb != nullptr) {
        // Copy the AABB's array data to device
        h_aabb->AABBMemcpyHtD();

        // Copy the AABB object to device
        CUDA_CHECK(
            hipMemcpy(d_aabb, h_aabb, sizeof(AABB), hipMemcpyHostToDevice));

        // Copy the entire mesh array to device
        if (d_mesh != nullptr && h_mesh != nullptr) {
            CUDA_CHECK(hipMemcpy(d_mesh, h_mesh,
                                  num_objects * sizeof(TriangleMesh),
                                  hipMemcpyHostToDevice));
        }

        // Copy the number of triangles to device
        if (d_num_triangles != nullptr && h_num_triangles != nullptr) {
            CUDA_CHECK(hipMemcpy(d_num_triangles, h_num_triangles,
                                  num_objects * sizeof(int),
                                  hipMemcpyHostToDevice));
        }
    }
}

// Copy data from device
__host__ void SceneManager::copyFromDevice() {
    // Copy rendered image from device
    CUDA_CHECK(hipMemcpy(h_image, d_image, width * height * sizeof(Vec3),
                          hipMemcpyDeviceToHost));
}

// Save image to file
__host__ void SceneManager::saveImage(const char* filename) {
    writeToPPM(filename, h_image, width, height);
    std::cout << "Image saved to " << filename << std::endl;
}

__host__ void SceneManager::renderBox() {
    // Get camera parameters from the stored reference
    CUDACameraParams camera_params = camera.CUDAparams();

    // Define grid and block dimensions
    dim3 block_dim(16, 16);
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x,
                  (height + block_dim.y - 1) / block_dim.y);

    // Launch kernel
    Timer timer;
    timer.start("Rendering Scene");
    renderBoxKernel<<<grid_dim, block_dim>>>(d_image, camera_params, d_aabb,
                                             num_objects);
    hipDeviceSynchronize();
    timer.stop();
}

__host__ void SceneManager::renderMesh() {
    // Get camera parameters from the stored reference
    CUDACameraParams camera_params = camera.CUDAparams();

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_materials), materials,
                                  sizeof(Material) * num_objects));

    // Define grid and block dimensions
    dim3 block_dim(32, 4);  // Adjusted for better warp scheduling / occupancy
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x,
                  (height + block_dim.y - 1) / block_dim.y);

    Timer timer;
    timer.start("Rendering Scene");
    renderMeshKernel<<<grid_dim, block_dim>>>(
        d_image, d_aabb, d_mesh, num_objects, d_num_triangles, camera_params);
    hipDeviceSynchronize();
    timer.stop();
    CUDA_CHECK(hipGetLastError());
}

__host__ void SceneManager::renderSpectralMesh(int samples_per_pixel) {
    // Get camera parameters from the stored reference
    CUDACameraParams camera_params = camera.CUDAparams();

    // Copy material data to constant memory
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_materials), materials,
                                  sizeof(Material) * num_objects));

    // Copies all the spectral data to constant memory
    initializeSpectra();

    // Define grid and block dimensions
    dim3 block_dim(16, 8);  // 128 threads per block
    dim3 grid_dim((width + block_dim.x - 1) / block_dim.x,
                  (height + block_dim.y - 1) / block_dim.y);

    Timer timer;
    timer.start("Rendering Spectral Scene");

    // Set seed here for now
    // unsigned int seed = static_cast<unsigned int>(time(nullptr)); // Random
    unsigned int seed = 1337;  // Fixed

    // Using signalling to print the progress
    // https://stackoverflow.com/a/20381924
    volatile int* h_progress;
    volatile int* d_progress;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc((void**)&h_progress, sizeof(int), hipHostMallocMapped);
    hipHostGetDevicePointer((void**)&d_progress, (int*)h_progress, 0);
    *h_progress = 0;

    renderSpectralMeshKernel<<<grid_dim, block_dim>>>(
        d_image, d_aabb, d_mesh, num_objects, d_num_triangles, camera_params,
        samples_per_pixel, seed, d_progress);

    // Copy the progress back to host
    int scanline_progress =
        (grid_dim.x * grid_dim.y + 16 - 1) / 16;  // Rounds up
    float current_progress = 0.0f;
    do {
        std::this_thread::sleep_for(
            std::chrono::milliseconds(100));  // lazy checker
        int val = *h_progress;
        float kern_progress =
            static_cast<float>(val) / static_cast<float>(scanline_progress);
        if (kern_progress - current_progress > 0.01f) {
            current_progress = kern_progress;
            printf("Progress: %2.1f%%\r", (kern_progress * 100));
            fflush(stdout);
        } else if (val >= scanline_progress - 1) {
            break;
        }
    } while (current_progress < 1.0f);
    printf("Progress: 100.0%%\n");


    hipDeviceSynchronize();
    timer.stop();
    CUDA_CHECK(hipGetLastError());

    // Free progress resources
    hipHostFree((void*)h_progress);
}

// Get AABB bounds for all objects
__host__ void SceneManager::getAABBBounds(Vec3& min_bounds, Vec3& max_bounds) {
    // Get the min and max bounds from the AABB object
    min_bounds = Vec3(h_aabb->h_minx[0], h_aabb->h_miny[0], h_aabb->h_minz[0]);
    max_bounds = Vec3(h_aabb->h_maxx[0], h_aabb->h_maxy[0], h_aabb->h_maxz[0]);
}