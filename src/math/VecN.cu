#include "hip/hip_runtime.h"
#include "VecN.cuh"
#include <cassert>
#include <stdexcept>


template<typename T>
VecN<T>::~VecN<T>() {
    delete[] pv;
}

template<typename T>
__global__ void kerAdd(T* oper1, T* oper2, T* res, int N) {
    // most boring thread mapping ever.
    unsigned int tid = blockDim.x*blockIdx.x + threadIdx.x;

    if (tid >= N) {
        return;
    }
    res[tid] = oper2[tid] + oper1[tid];
}

template<typename T>
VecN<T>* VecN<T>::deviceAdd(VecN<T>* vec2) {

    hipError_t lastError = hipGetLastError();

    if (this->N != vec2->N) {
        throw std::runtime_error(
                "[ERROR, VecN::deviceAdd()]:  Cannot add vectors of mismatched length!"
                );
    }


    // allocate memory for device oper1, oper2, res.

    T* devOper1;
    T* devOper2;
    T* devRes;

    int vecBytes = N*sizeof(T);
    hipMalloc(&devOper1, vecBytes);
    hipMalloc(&devOper2, vecBytes);
    hipMalloc(&devRes, vecBytes);

    lastError = hipGetLastError();
    if (lastError != hipSuccess) {
        throw std::runtime_error("[ERROR, VecN::deviceAdd()]: Failed to allocate memory!");
    }

    // transfer this to device, transfer vec2 to device.

    hipMemcpy(devOper1, this->pv, vecBytes, hipMemcpyHostToDevice);
    hipMemcpy(devOper2, vec2->pv, vecBytes, hipMemcpyHostToDevice);

    lastError = hipGetLastError();
    if (lastError != hipSuccess) {
        throw std::runtime_error("[ERROR, VecN::deviceAdd()]: Failed to transfer host memory to device!");
    }

    // launch kernel.

    dim3 blockDim = dim3( BLOCK_SIZE, 1, 1 );
    dim3 gridDim = dim3( (N+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1 );

    kerAdd<<<gridDim, blockDim>>>(devOper1, devOper2, devRes, N);

    // transfer devRes to host.
    T* pv = (T*) malloc(vecBytes);
    hipMemcpy(pv, devRes, vecBytes, hipMemcpyDeviceToHost);

    VecN<T>* res = new VecN<T>(N, pv);

    // free all GPU memory.
    hipFree(devOper1);
    hipFree(devOper2);
    hipFree(devRes);
    lastError = hipGetLastError();
    if (lastError != hipSuccess) {
        throw std::runtime_error("[ERROR, VecN::deviceAdd()]: Failed to free device memory!");
    }

    return res;
}

template class VecN<float>;
template class VecN<int>;
template __global__ void kerAdd(float*, float*, float*, int);
